#include "hip/hip_runtime.h"
#pragma once

#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <hiprand/hiprand_kernel.h>

#include "mesh.h"
#include "utils.h"
#include "gpu_traverse.cuh"
#include "mesh_sampler.cuh"

CUDA_GLOBAL void mesh_sample_surface_uniform_entry(
    const glm::vec3 *i_vertices,
    const Face *i_faces,
    const float *i_face_weights_prefix_sum,
    int n_faces,
    hiprandState *io_rand_states,
    glm::vec3 *o_out_points,
    int n_points
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_points) return;

    hiprandState local_state = io_rand_states[idx];

    // Sample a face based on the face weights
    float r = hiprand_uniform(&local_state);
    int face_idx = thrust::upper_bound(thrust::seq, i_face_weights_prefix_sum, i_face_weights_prefix_sum + n_faces, r) - i_face_weights_prefix_sum;
    face_idx = min(face_idx, n_faces - 1);

    Face face = i_faces[face_idx];
    glm::vec3 v0 = i_vertices[face.v1];
    glm::vec3 v1 = i_vertices[face.v2];
    glm::vec3 v2 = i_vertices[face.v3];

    // Sample a point uniformly within the triangle
    float u = hiprand_uniform(&local_state);
    float v = hiprand_uniform(&local_state);

    if (u + v > 1.0f) {
        u = 1.0f - u;
        v = 1.0f - v;
    }

    glm::vec3 sampled_point = (1 - u - v) * v0 + u * v1 + v * v2;

    o_out_points[idx] = sampled_point;
    io_rand_states[idx] = local_state;
}